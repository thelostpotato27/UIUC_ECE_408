#include "hip/hip_runtime.h"
__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<len){out[i] = in1[i] + in2[i];}
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int size = inputLength * sizeof(float);
  float *in1_d, *in2_d, *in3_d;
  hipMalloc((void**) &in1_d, size);
  hipMalloc((void**) &in2_d, size);
  hipMalloc((void**) &in3_d, size);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(in1_d, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(in2_d, hostInput2, size, hipMemcpyHostToDevice);
  //@@ Copy memory to the GPU here

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  dim3 DimGrid(ceil(size/256.0),1,1);
  dim3 DimBlock(256,1,1);
  //@@ Initialize the grid and block dimensions here

  wbTime_start(Compute, "Performing CUDA computation");
  vecAdd<<<DimGrid, DimBlock>>>(in1_d,in2_d,in3_d,size);
  //@@ Launch the GPU Kernel here

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, in3_d, size, hipMemcpyDeviceToHost);
  //@@ Copy the GPU memory back to the CPU here

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(in1_d);
  hipFree(in2_d);
  hipFree(in3_d);
  //@@ Free the GPU memory here

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}