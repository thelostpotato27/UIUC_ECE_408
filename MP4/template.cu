#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 8
#define MASK_WIDTH 5
//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int mx = blockIdx.x * TILE_WIDTH + tx;
    int my = blockIdx.y * TILE_WIDTH + ty;
    int mz = blockIdx.z * TILE_WIDTH + tz;
    int xl = mx - (MASK_WIDTH-1)/2;
    int yl = my - (MASK_WIDTH-1)/2;
    int zl = mz - (MASK_WIDTH-1)/2;
    int xr = mx + (MASK_WIDTH-1)/2;
    int yr = my + (MASK_WIDTH-1)/2;
    int zr = mz + (MASK_WIDTH-1)/2;

    __shared__ float shared_arr[TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1];

    if (xl >= 0 && xr < x_size && yl >= 0 && yr < y_size && zl >= 0 && zr < z_size){
        shared_arr[tz][ty][tx] = input[zl * (y_size * x_size) + yl * (x_size) + xl];
    }else{
        shared_arr[tz][ty][tx] = 0.0;
    }

    __syncthreads();

    if (tx < TILE_WIDTH && ty < TILE_WIDTH && tz < TILE_WIDTH && mx < x_size && my < y_size && mz < z_size){
        float Pvalue = 0;
        for (int i = 0; i < MASK_WIDTH; i++){
            for (int j = 0; j < MASK_WIDTH; j++){
                for (int k = 0; k < MASK_WIDTH; k++){
                    Pvalue += M[i][j][k] * shared_arr[tz + i][ty + j][tx + k];
                }
            }
        }
        output[mz * (y_size * x_size) + my * (x_size) + mx] = Pvalue;
    }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions

  hipMalloc((void **)&deviceInput, z_size * y_size * x_size * sizeof(float));
  hipMalloc((void **)&deviceOutput, z_size * y_size * x_size * sizeof(float));

  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], z_size * y_size * x_size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M), hostKernel, MASK_WIDTH * MASK_WIDTH * MASK_WIDTH * sizeof(float));

  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil(x_size/double(TILE_WIDTH)),ceil(y_size/double(TILE_WIDTH)), ceil(z_size/double(TILE_WIDTH)));
  dim3 DimBlock(TILE_WIDTH + MASK_WIDTH - 1,TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1);

  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(&HostOutput[3], deviceOutput, z_size * y_size * x_size * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
